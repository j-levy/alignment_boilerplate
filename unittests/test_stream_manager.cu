#include "hip/hip_runtime.h"
#include <albp/doctest.hpp>
#include <albp/stream_manager.hpp>
#include <albp/memory.hpp>

using namespace albp;

namespace std {
  template<typename T, typename U>
  std::ostream& operator<< (std::ostream& os, const std::pair<T,U> &value) {
    os << "{" << value.first << "," << value.second << "}";
    return os;
  }
}


__global__ void gpu_test_kernel(double *const data, const int count){
  for (int di = blockIdx.x * blockDim.x + threadIdx.x; di < count; di += blockDim.x * gridDim.x){
    for(int i=0;i<20;i++)
      data[di] = sin(data[di]);
  }
}


TEST_CASE("Stream Manager"){
  const size_t work_items = 50'000;
  std::vector<double> data_vec;
  data_vec.reserve(work_items);

  for(size_t i=0;i<work_items;i++)
    data_vec.push_back(i%5);

  struct GPUStorage {
    double *data   = nullptr;
  };

  struct StreamStorage {
    double *d_data = nullptr;
  };

  const std::vector<int> gpu_ids = {0};
  const int streams_per_gpu  = 6;
  const int chunks_per_gpu   = 6;
  const int thread_pool_size = 6;

  GPUManager gpu_manager(gpu_ids, streams_per_gpu, thread_pool_size);

  const GPUSetupFunc<GPUStorage> gpu_setup = [&](const int gpu_id, const RangePair range, const size_t chunks, const size_t max_chunk_size) -> GPUStorage {
    (void)gpu_id; //Suppress unused variable warning
    GPUStorage gpu_storage;
    gpu_storage.data = PageLockedMalloc<double>(range.size());
    memcpy(gpu_storage.data, data_vec.data(), range.size()*sizeof(double));
    return gpu_storage;
  };

  const StreamSetupFunc<GPUStorage,StreamStorage> stream_setup = [&](const GPUStorage &gpu_storage, const hipStream_t stream, const size_t max_chunk_size){
    StreamStorage temp;
    temp.d_data = DeviceMalloc<double>(max_chunk_size);
    return temp;
  };

  const TaskFunc<GPUStorage,StreamStorage> task_func = [](const GPUStorage &gpu_storage, const StreamStorage &stream_storage, const hipStream_t stream, const RangePair range) {
    ALBP_CUDA_ERROR_CHECK(hipMemcpyAsync(stream_storage.d_data, gpu_storage.data+range.begin, range.size()*sizeof(double), hipMemcpyHostToDevice, stream));

    gpu_test_kernel<<<200,128,0,stream>>>(stream_storage.d_data, range.size());
    ALBP_CUDA_ERROR_CHECK(hipGetLastError());

    ALBP_CUDA_ERROR_CHECK(hipMemcpyAsync(gpu_storage.data+range.begin, stream_storage.d_data, range.size()*sizeof(double), hipMemcpyDeviceToHost, stream));
  };

  const TaskFinishFunc<GPUStorage,StreamStorage> task_finish = [&](const GPUStorage &gpu_storage, const StreamStorage &stream_storage, const hipStream_t stream, const RangePair range){};

  const StreamFinishFunc<GPUStorage,StreamStorage> stream_finish = [&](const GPUStorage &gpu_storage, StreamStorage &stream_storage, const hipStream_t stream, const size_t max_chunk_size){
    ALBP_CUDA_ERROR_CHECK(hipFree(stream_storage.d_data));
  };

  const GPUFinishFunc<GPUStorage> gpu_finish = [&](const int gpu_id, GPUStorage &gpu_storage){};

  const auto storage = TaskWork(
    gpu_manager,
    work_items,
    chunks_per_gpu,
    gpu_setup,
    stream_setup,
    task_func,
    task_finish,
    stream_finish,
    gpu_finish
  );

  ALBP_CUDA_ERROR_CHECK(hipDeviceSynchronize());

  bool good = true;
  for(int wi=0;wi<work_items;wi++){
    double answer = wi%5;
    for(int i=0;i<20;i++)
      answer = std::sin(answer);
    good &= (answer==doctest::Approx(storage.at(0).gpu_storage.data[wi]));
  }

  CHECK(good);

  for(auto &kv: storage){
    ALBP_CUDA_ERROR_CHECK(hipHostFree(kv.second.gpu_storage.data));
  }
}
